/* 
 * File:   data.cu
 * Author: Rik Schaaf aka CC007 <coolcat007.nl>
 *
 * Created on May 27, 2015, 9:03 PM
 */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../../include/data.h"
#include "../../include/safemem.h"

void cudaMallocData(Data *c, int iter, int p) {
    if (iter > 0) {
        int i;
        hipMalloc((void**) c, p * sizeof (Data));
        for (i = 0; i < p; i++) {
            Data helper_d;
            helper_d.length = iter;
            hipMalloc((void**) &(helper_d.x), iter * sizeof (double));
            hipMalloc((void**) &(helper_d.dx), iter * sizeof (double));
            hipMalloc((void**) &(helper_d.y), iter * sizeof (double));
            hipMalloc((void**) &(helper_d.dy), iter * sizeof (double));
            hipMalloc((void**) &(helper_d.delta), iter * sizeof (double));
            hipMalloc((void**) &(helper_d.phi), iter * sizeof (double));
            hipMemcpy(&((*c)[i]), &helper_d, sizeof (Data), hipMemcpyHostToDevice);
        }
    }
}

void cudaMemcpyMap(Map *dst_m, Map *src_m, hipMemcpyKind kind) {
    Map helper_m;
    if (kind == hipMemcpyDeviceToHost) {
        hipMemcpy(&helper_m, src_m, sizeof (Map), hipMemcpyDeviceToHost);
        hipMemcpy(dst_m->A, helper_m.A, helper_m.length * sizeof (double), kind);
        hipMemcpy(dst_m->x, helper_m.x, helper_m.length * sizeof (int), kind);
        hipMemcpy(dst_m->dx, helper_m.dx, helper_m.length * sizeof (int), kind);
        hipMemcpy(dst_m->y, helper_m.y, helper_m.length * sizeof (int), kind);
        hipMemcpy(dst_m->dy, helper_m.dy, helper_m.length * sizeof (int), kind);
        hipMemcpy(dst_m->delta, helper_m.delta, helper_m.length * sizeof (int), kind);
        hipMemcpy(dst_m->phi, helper_m.phi, helper_m.length * sizeof (int), kind);
    } else if (kind == hipMemcpyHostToDevice) {
        hipMemcpy(&helper_m, dst_m, sizeof (Map), hipMemcpyDeviceToHost);
        hipMemcpy(helper_m.A, src_m->A, helper_m.length * sizeof (double), kind);
        hipMemcpy(helper_m.x, src_m->x, helper_m.length * sizeof (int), kind);
        hipMemcpy(helper_m.dx, src_m->dx, helper_m.length * sizeof (int), kind);
        hipMemcpy(helper_m.y, src_m->y, helper_m.length * sizeof (int), kind);
        hipMemcpy(helper_m.dy, src_m->dy, helper_m.length * sizeof (int), kind);
        hipMemcpy(helper_m.delta, src_m->delta, helper_m.length * sizeof (int), kind);
        hipMemcpy(helper_m.phi, src_m->phi, helper_m.length * sizeof (int), kind);
    } else {
        fprintf(stderr, "DeviceToDevice is not yet supported for maps!\n");
        getchar();
        exit(EXIT_FAILURE);
    }

}