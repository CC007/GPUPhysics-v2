/* 
 * File:   map.cu
 * Author: Rik Schaaf aka CC007 <coolcat007.nl>
 *
 * Created on May 27, 2015, 9:09 PM
 */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "../../include/map.h"
#include "../../include/safemem.h"

void cudaMallocMap(Map *mapPointer, int rowCount) {
	Map hostHelperMap;
	Map devHelperMap;
	if (safeMalloc((void**) &hostHelperMap, 1, sizeof (struct _Map))) {
		eprintf("The map could not be allocated on the host");
	}
	if (safeCudaMalloc((void**) &devHelperMap, 1, sizeof (struct _Map))) {
		eprintf("The map could not be allocated on the device");
	}
	hostHelperMap->length = rowCount;
	if (rowCount > 0) {
		int mallocFailed = 0;
		mallocFailed += safeCudaCalloc((void**) &(hostHelperMap->A), rowCount, sizeof (double));
		mallocFailed += safeCudaCalloc((void**) &(hostHelperMap->x), rowCount, sizeof (int));
		mallocFailed += safeCudaCalloc((void**) &(hostHelperMap->dx), rowCount, sizeof (int));
		mallocFailed += safeCudaCalloc((void**) &(hostHelperMap->y), rowCount, sizeof (int));
		mallocFailed += safeCudaCalloc((void**) &(hostHelperMap->dy), rowCount, sizeof (int));
		mallocFailed += safeCudaCalloc((void**) &(hostHelperMap->delta), rowCount, sizeof (int));
		mallocFailed += safeCudaCalloc((void**) &(hostHelperMap->phi), rowCount, sizeof (int));
		if (mallocFailed) {
			eprintf("The map's contents could not be allocated");
		}
		if (safeCudaMemcpyHtD(&devHelperMap, &hostHelperMap, 1, sizeof (struct _Map))) {
			eprintf("The map's contents could not be made available in device memory (temporary host map memcpy failed)");
		}
		if (safeFree((void**) &hostHelperMap)) {
			wprintf("The temporary host map's contents could not be freed");
		}
		*mapPointer = devHelperMap;
	}
}

void cudaFreeMap(Map *mapPointer) {
	Map devHelperMap = *mapPointer;
	Map hostHelperMap;
	if (safeMalloc((void**) &hostHelperMap, 1, sizeof (struct _Map))) {
		wprintf("The map's contents could not be accessed (temporary host map alloc failed)");
		return;
	}
	if (safeCudaMemcpyDtH(&hostHelperMap, &devHelperMap, 1, sizeof (struct _Map))) {
		wprintf("The map's contents could not be accessed (temporary host map memcpy failed)");
		return;
	}
	int freeFailed = 0;
	if (hostHelperMap->length > 0) {
		freeFailed += safeCudaFree((void**) &(hostHelperMap->A));
		freeFailed += safeCudaFree((void**) &(hostHelperMap->x));
		freeFailed += safeCudaFree((void**) &(hostHelperMap->dx));
		freeFailed += safeCudaFree((void**) &(hostHelperMap->y));
		freeFailed += safeCudaFree((void**) &(hostHelperMap->dy));
		freeFailed += safeCudaFree((void**) &(hostHelperMap->delta));
		freeFailed += safeCudaFree((void**) &(hostHelperMap->phi));
	}
	if (freeFailed) {
		wprintf("The map's contents could not be freed");
	}
	if (safeFree((void**) &hostHelperMap)) {
		wprintf("The temporary host map's contents could not be freed");
	}
	if (safeCudaFree((void**) mapPointer)) {
		wprintf("The map could not be freed");
	}
}