/* 
 * File:   map.cu
 * Author: Rik Schaaf aka CC007 <coolcat007.nl>
 *
 * Created on May 27, 2015, 9:09 PM
 */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "../../include/map.h"
#include "../../include/safemem.h"
#include "../../include/extendedio.h"

void cudaMallocMap(Map *mapPointer, int rowCount) {
	Map hostHelperMap;
	Map devHelperMap;
	if (safeMalloc((void**) &hostHelperMap, 1, sizeof (struct _Map))) {
		eprintf("The map could not be allocated on the host");
	}
	if (safeCudaMalloc((void**) &devHelperMap, 1, sizeof (struct _Map))) {
		eprintf("The map could not be allocated on the device");
	}
	hostHelperMap->length = rowCount;
	if (rowCount > 0) {
		int mallocFailed = 0;
		mallocFailed += safeCudaCalloc((void**) &(hostHelperMap->A), rowCount, sizeof (double));
		mallocFailed += safeCudaCalloc((void**) &(hostHelperMap->x), rowCount, sizeof (int));
		mallocFailed += safeCudaCalloc((void**) &(hostHelperMap->dx), rowCount, sizeof (int));
		mallocFailed += safeCudaCalloc((void**) &(hostHelperMap->y), rowCount, sizeof (int));
		mallocFailed += safeCudaCalloc((void**) &(hostHelperMap->dy), rowCount, sizeof (int));
		mallocFailed += safeCudaCalloc((void**) &(hostHelperMap->delta), rowCount, sizeof (int));
		mallocFailed += safeCudaCalloc((void**) &(hostHelperMap->phi), rowCount, sizeof (int));
		if (mallocFailed) {
			eprintf("The map's contents could not be allocated");
		}
		if (safeCudaMemcpyHtD(devHelperMap, hostHelperMap, 1, sizeof (struct _Map))) {
			eprintf("The map's contents could not be made available in device memory (temporary host map memcpy failed)");
		}
		if (safeFree((void**) &hostHelperMap)) {
			wprintf("The temporary host map's contents could not be freed");
		}
		*mapPointer = devHelperMap;
	}
}

void cudaFreeMap(Map *mapPointer) {
	Map devHelperMap = *mapPointer;
	Map hostHelperMap;
	if (safeMalloc((void**) &hostHelperMap, 1, sizeof (struct _Map))) {
		wprintf("The map's contents could not be accessed (temporary host map alloc failed)");
		return;
	}
	if (safeCudaMemcpyDtH(&hostHelperMap, &devHelperMap, 1, sizeof (struct _Map))) {
		wprintf("The map's contents could not be accessed (temporary host map memcpy failed)");
		return;
	}
	int freeFailed = 0;
	if (hostHelperMap->length > 0) {
		freeFailed += safeCudaFree((void**) &(hostHelperMap->A));
		freeFailed += safeCudaFree((void**) &(hostHelperMap->x));
		freeFailed += safeCudaFree((void**) &(hostHelperMap->dx));
		freeFailed += safeCudaFree((void**) &(hostHelperMap->y));
		freeFailed += safeCudaFree((void**) &(hostHelperMap->dy));
		freeFailed += safeCudaFree((void**) &(hostHelperMap->delta));
		freeFailed += safeCudaFree((void**) &(hostHelperMap->phi));
	}
	if (freeFailed) {
		wprintf("The map's contents could not be freed");
	}
	if (safeFree((void**) &hostHelperMap)) {
		wprintf("The temporary host map's contents could not be freed");
	}
	if (safeCudaFree((void**) mapPointer)) {
		wprintf("The map could not be freed");
	}
}

void cudaMemcpyMap(Map destinationMap, Map sourceMap, hipMemcpyKind kind) {
	Map hostHelperMap;
	if (safeMalloc((void**) &hostHelperMap, 1, sizeof (struct _Map))) {
		eprintf("The map could not be copied (temporary host map alloc failed)");
	}
	int memcpyFailed = 0;
	if (kind == hipMemcpyDeviceToHost) {
		if (safeCudaMemcpyDtH(hostHelperMap, sourceMap, 1, sizeof (struct _Map))) {
			eprintf("The map could not be copied (source map pointers couldn't be accessed)");
		}
		memcpyFailed += safeCudaMemcpyDtH(destinationMap->A, hostHelperMap->A, hostHelperMap->length, sizeof (double));
		memcpyFailed += safeCudaMemcpyDtH(destinationMap->x, hostHelperMap->x, hostHelperMap->length, sizeof (int));
		memcpyFailed += safeCudaMemcpyDtH(destinationMap->dx, hostHelperMap->dx, hostHelperMap->length, sizeof (int));
		memcpyFailed += safeCudaMemcpyDtH(destinationMap->y, hostHelperMap->y, hostHelperMap->length, sizeof (int));
		memcpyFailed += safeCudaMemcpyDtH(destinationMap->dy, hostHelperMap->dy, hostHelperMap->length, sizeof (int));
		memcpyFailed += safeCudaMemcpyDtH(destinationMap->delta, hostHelperMap->delta, hostHelperMap->length, sizeof (int));
		memcpyFailed += safeCudaMemcpyDtH(destinationMap->phi, hostHelperMap->phi, hostHelperMap->length, sizeof (int));
	} else if (kind == hipMemcpyHostToDevice) {
		if (safeCudaMemcpyDtH(hostHelperMap, destinationMap, 1, sizeof (struct _Map))) {
			eprintf("The map could not be copied (destination map pointers couldn't be accessed)");
		}
		memcpyFailed += safeCudaMemcpyHtD(hostHelperMap->A, sourceMap->A, hostHelperMap->length,sizeof (double));
		memcpyFailed += safeCudaMemcpyHtD(hostHelperMap->x, sourceMap->x, hostHelperMap->length, sizeof (int));
		memcpyFailed += safeCudaMemcpyHtD(hostHelperMap->dx, sourceMap->dx, hostHelperMap->length, sizeof (int));
		memcpyFailed += safeCudaMemcpyHtD(hostHelperMap->y, sourceMap->y, hostHelperMap->length, sizeof (int));
		memcpyFailed += safeCudaMemcpyHtD(hostHelperMap->dy, sourceMap->dy, hostHelperMap->length, sizeof (int));
		memcpyFailed += safeCudaMemcpyHtD(hostHelperMap->delta, sourceMap->delta, hostHelperMap->length, sizeof (int));
		memcpyFailed += safeCudaMemcpyHtD(hostHelperMap->phi, sourceMap->phi, hostHelperMap->length, sizeof (int));
	} else {
		eprintf("DeviceToDevice is not yet supported for maps!\n");
	}
	if(memcpyFailed){
			eprintf("The map could not be copied (copying the content of the map failed)");
	}
}